#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"
#include "math_functions.h"
#include "hipblas.h"
#include <thread>
#include <xthreads.h>

#define DLLEXPORT __declspec(dllexport)


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

//
//float DecodeFloatRGBA(float4 enc)
//{
//    float4 kDecodeDot = float4();
//    kDecodeDot.x = 1.0f;
//    kDecodeDot.y = 1 / 255.0;
//    kDecodeDot.z = 1 / 65025.0;
//    kDecodeDot.w = 1 / 16581375.0;
//
//    float depth = hipblasSdot(1, (float*)&enc, 4, (float*)&kDecodeDot, 4);
//    return depth;
//}

// future
__global__ void voxelizedDiff(BYTE* g_data, unsigned int* g_dataSrc, float frontDepth, float backDepth, unsigned int size, unsigned int scaler) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int xSrc = x * scaler;
    int ySrc = y * scaler;
    int index = y * size + x;
    bool isLit = true;
    bool isShadow = true;
    int originSize = size * scaler;
    for (int v = 0; v < scaler; v++) {
        for (int u = 0; u < scaler; u++) {
            int vSrc = ySrc + v;
            int uSrc = xSrc + u;
            int indexSrc = vSrc * originSize + uSrc;
            unsigned int* src = g_dataSrc + indexSrc;
            BYTE* srcChannel = (BYTE*)src;
            float4 rgba;
            rgba.x = src[0] / 255.0f;
            rgba.y = src[1] / 255.0f;
            rgba.z = src[2] / 255.0f;
            rgba.w = src[3] / 255.0f;
            float4 kDecodeDot = float4();
            kDecodeDot.x = 1.0f;
            kDecodeDot.y = 1 / 255.0;
            kDecodeDot.z = 1 / 65025.0;
            kDecodeDot.w = 1 / 16581375.0;

            float depth = rgba.x * kDecodeDot.x + rgba.y * kDecodeDot.y + rgba.z * kDecodeDot.z + rgba.w * kDecodeDot.w; //hipblasSdot(4, (float*)&rgba, 0, (float*)&kDecodeDot, 0);
            isLit &= abs((int)src - 255) < 20;
            isShadow &= (int)src < 20;
        }
    }
}

// blockSize = 64 * 64   threadDim(32 * 32,1)= (1024,1)= (32,32)
__global__ void voxelizedSample(BYTE* src, BYTE* dst, unsigned int lv4VoxelSize, unsigned int lv4PixelPerVoxel) {

    int uPixelTarget = threadIdx.x;
    int vPixelTarget = threadIdx.x;
}

const int SCALER = 2;

// g_data 2048*2048   g_dataSrc 4096 Texture2DArray subArray
__global__ void voxelizedSample_kernel(BYTE* g_data, BYTE* g_dataSrc, unsigned int size , unsigned int scaler)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int xSrc = x * scaler;
    int ySrc = y * scaler;
    int index = y * size + x;
    bool isLit = true;
    bool isShadow = true;
    
    int originSize = size * scaler;
    for (int v = 0; v < scaler; v++) {
        for (int u = 0; u < scaler; u++) {
            int vSrc = ySrc + v;
            int uSrc = xSrc + u;
            int indexSrc = vSrc * originSize + uSrc;
            BYTE src = g_dataSrc[indexSrc];
            isLit &= abs((int)src - 255) < 20;
            isShadow &= (int)src < 20;
        }
    }
    bool isIntersected = !isLit && !isShadow;
    g_data[index] = (BYTE)((isLit ? 255 : 0) + (isIntersected ? 128 : 0));
    
    
}

// Lv4 summary to lv3 support 2048 to 256
__global__ void voxelizedSample3D_kernel(BYTE* g_data, BYTE* g_dataSrc, unsigned int size, unsigned int scaler) {
    g_data[threadIdx.x] = 0;
}


#define CHECK_ERR(status)   \
    if (cudaStatus != status) {    \
        fprintf(stderr, "hipSetDevice failed!  errorCode: %i ?", (int)status); \
        goto Error; \
    }   


static bool memPoolEnabled;
// target device memory pool
static std::vector<BYTE*>* poolTarget;

// origin device memory pool
static std::vector<BYTE*>* poolOrigin;

static int TARGET_BUFFER_POOL_SIZE = 16;
static int ORIGIN_BUFFER_POOL_SIZE = 64;

static int targetBufferInUse = 0;
static int targetBufferFree = 0;
static int originBufferInUse = 0;
static int originBufferFree = 0;
static _Mtx_t bufferMtx;

static int nThreadNum = 16;

void lockBufferMtx() {
    _Mtx_lock(bufferMtx);
}
void unlockBufferMtx() {
    _Mtx_unlock(bufferMtx);
}
bool hasTargetBuff() {
    lockBufferMtx();
    bool b = targetBufferFree > 0;
    unlockBufferMtx();
    return b;
}

bool hasOriginBuff() {
    lockBufferMtx();
    bool b = originBufferFree > 0;
    unlockBufferMtx();
    return b;
}

BYTE* getTargetBuffer(BYTE** ptr) {
    lockBufferMtx();
    if (targetBufferFree > 0) {
        *ptr = *poolTarget->begin();
        poolTarget->erase(poolTarget->begin());
        targetBufferInUse++;
        targetBufferFree--;
    }
    unlockBufferMtx();
    return *ptr;
}
void reclaimTargetBuffer(BYTE** ptr) {
    lockBufferMtx();
    poolTarget->push_back(*ptr);
    targetBufferInUse--;
    targetBufferFree++;
    unlockBufferMtx();
}
void reclaimOriginBuffer(BYTE** ptr) {
    lockBufferMtx();
    poolOrigin->push_back(*ptr);
    originBufferInUse--;
    originBufferFree++;
    unlockBufferMtx();
}
BYTE* getOriginBuffer(BYTE** ptr) {
    lockBufferMtx();
    if (originBufferFree > 0) {
        *ptr = *poolOrigin->begin();
        poolOrigin->erase(poolOrigin->begin());
        originBufferInUse++;
        originBufferFree--;
    }
    unlockBufferMtx();
    return *ptr;
}

extern "C" {
    DLLEXPORT void Init(unsigned int targetBufferPoolSize, unsigned int originBufferPoolSize,  unsigned int targetSize, unsigned int scaler = SCALER, unsigned int threadNum = 16) {
        _Mtx_init(&bufferMtx, 0);
        TARGET_BUFFER_POOL_SIZE = targetBufferPoolSize;
        ORIGIN_BUFFER_POOL_SIZE = originBufferPoolSize;
        poolTarget = new std::vector<BYTE*>();
        poolOrigin = new std::vector<BYTE*>();
        for (int i = 0; i < targetBufferPoolSize; i++) {
            BYTE* targetTex;
            hipMalloc<BYTE>(&targetTex, targetSize * targetSize);
            poolTarget->push_back(targetTex);
            
        }
        for (int i = 0; i < originBufferPoolSize; i++) {
            BYTE* originTex;
            hipMalloc<BYTE>(&originTex, targetSize * targetSize * scaler * scaler);
            poolOrigin->push_back(originTex);
        }
        targetBufferFree = targetBufferPoolSize;
        originBufferFree = originBufferPoolSize;
        nThreadNum = threadNum;
        memPoolEnabled = true;
    }

    DLLEXPORT void Close() {
        for (auto i = poolTarget->begin(), c = poolTarget->end(); i != c; i++) {
            auto value = *i;
            hipFree(value);
        }
        for (auto i = poolOrigin->begin(), c = poolOrigin->end(); i != c; i++) {
            auto value = *i;
            hipFree(value);
        }
    }

    DLLEXPORT hipError_t Downsample(BYTE* targetTex, BYTE* originTex, unsigned int targetSize, unsigned int scaler = SCALER) {
        BYTE* dev_targetTex;
        BYTE* dev_originTex;
        
        hipError_t cudaStatus;
        cudaStatus = hipSetDevice(0);
        CHECK_ERR(cudaStatus);
        if (memPoolEnabled) {
            while (!hasTargetBuff())
                Sleep(30);
            getTargetBuffer(&dev_targetTex);
            while (!hasOriginBuff())
                Sleep(30);
            getOriginBuffer(&dev_originTex);
            
        }
        else {
            cudaStatus = hipMalloc<BYTE>(&dev_targetTex, targetSize * targetSize);
            CHECK_ERR(cudaStatus);
            cudaStatus = hipMalloc<BYTE>(&dev_originTex, targetSize * targetSize * scaler * scaler);
            CHECK_ERR(cudaStatus);
        }
        

        unsigned int threadNum = min(nThreadNum, targetSize);
        dim3 threads = dim3(threadNum, threadNum);
        dim3 blocks = dim3(targetSize / threads.x, targetSize / threads.y);

        // create cuda event handles
        hipEvent_t start, stop;
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));

        hipStream_t streamId;
        hipStreamCreate(&streamId);

        StopWatchInterface* timer = NULL;
        sdkCreateTimer(&timer);
        sdkResetTimer(&timer);
        checkCudaErrors(hipDeviceSynchronize());
        float gpu_time = 0.0f;
        // asynchronously issue work to the GPU (all to stream 0)
        sdkStartTimer(&timer);
        hipEventRecord(start, streamId);

        cudaStatus = hipMemcpyAsync(dev_originTex, originTex, targetSize * targetSize * scaler * scaler, hipMemcpyHostToDevice, streamId);

        CHECK_ERR(cudaStatus);
        voxelizedSample_kernel << <blocks, threads, 0, streamId >> > (dev_targetTex, dev_originTex, targetSize, scaler);
        CHECK_ERR(cudaStatus);

        cudaStatus = hipMemcpyAsync(targetTex, dev_targetTex, targetSize * targetSize, hipMemcpyDeviceToHost, streamId);
        hipEventRecord(stop, streamId);
        sdkStopTimer(&timer);

        //cudaStatus = hipDeviceSynchronize();
        
        // have CPU do some work while waiting for stage 1 to finish
        unsigned long int counter = 0;

        //while (hipEventQuery(stop) == hipErrorNotReady)
        //{
        //    counter++;
        //}
        //cudaStatus = hipEventSynchronize(stop);
        
        while (hipStreamQuery(streamId) != hipSuccess) {
            counter++;
        }
        // hipStreamWaitEvent(streamId, stop, 0);
        // hipStreamSynchronize(streamId);
        hipStreamDestroy(streamId);
        checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

        // print the cpu and gpu times
        printf("time spent executing by the GPU: %.2f\n", gpu_time);
        printf("time spent by CPU in CUDA calls: %.2f\n", sdkGetTimerValue(&timer));
        printf("CPU executed %lu iterations while waiting for GPU to finish\n", counter);

        // release resources
        checkCudaErrors(hipEventDestroy(start));
        checkCudaErrors(hipEventDestroy(stop));

        if (memPoolEnabled) {
            reclaimTargetBuffer(&dev_targetTex);
            reclaimOriginBuffer(&dev_originTex);
        }
        else {
            hipFree(dev_targetTex);
            hipFree(dev_originTex);
        }
        return cudaStatus;


    Error:
        if (memPoolEnabled) {
            reclaimTargetBuffer(&dev_targetTex);
            reclaimOriginBuffer(&dev_originTex);
        }
        else {
            hipFree(dev_targetTex);
            hipFree(dev_originTex);
        }
        return cudaStatus;
    }

    DLLEXPORT void* AllocMem(size_t size) {
        BYTE* ptr = (BYTE*) malloc(size);
        //hipMallocManaged<BYTE>(&ptr, size);
        return ptr;

    }

    DLLEXPORT void FreeMem(void* ptr) {
        free(ptr);
    }

    DLLEXPORT BYTE* GetSubArray(BYTE* ptr, INT64 start, INT64 length) {
        return ptr + start;
    }

}

void printDeviceInfo() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    int device;
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Device %d has compute capability %d.%d.\n",
            device, deviceProp.major, deviceProp.minor);
    }
}

int main1(){

    printDeviceInfo();

    int targetSize = 4096;
    BYTE* data = (BYTE*)malloc(targetSize * targetSize);
    //hipHostMalloc(&data, targetSize * targetSize);
    BYTE* originData = (BYTE*)malloc(targetSize * targetSize * 4);
    //hipHostMalloc(&originData, targetSize * targetSize * 4);
    //memset(data, (BYTE)0, targetSize * targetSize);
    int memSetValue = 0;
    BYTE oneByte[4] = { 255,255,255,255 };
    memcpy(&memSetValue, &oneByte, 4);
    memset(data, memSetValue,  1024 * 1024 / 4);
    memset(originData, memSetValue, targetSize * targetSize / 4);
    Init(8, 16, targetSize, 2);
    for (int i = 0; i < 512; i++) {
        Downsample(data, originData, targetSize, 2);
    }
    Close();
    for (int i = 0; i < 64; i++) {
        BYTE a = data[i * 64];
        printf("$$ a: %i", a);
    }
    return 0;
}

int main2()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
